#include "hip/hip_runtime.h"
#include <framework.cuh>

__device__ int getNextEmptyLocation(short *matrix, int *loc_row, int *loc_col)
{
    int i, j;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            if (matrix[i * N + j] == 0)
            {
                *loc_row = i;
                *loc_col = j;
                return 0;
            }
        }
    }
    return 1;
}

__device__ int isOK(short *matrix, int r, int c, int val)
{
    int i;
    for (i = 0; i < N; i++)
    {
        if (matrix[r * N + i] == val)
            return 0;
        if (matrix[i * N + c] == val)
            return 0;
    }
    return 1;
}

__device__ void createLatinSquare(short *matrix, int *ss, short *answer)
{
    size_t sp = 0;
    int x, r, c, retVal;

start:
    retVal = getNextEmptyLocation(matrix, &r, &c);
    if (retVal == 1)
    {
        // call test_function
        //test_function(matrix, answer);

        // popup variables in reverse order
        sp--;
        c = ss[sp];
        sp--;
        r = ss[sp];
        sp--;
        x = ss[sp];

        // return to the code
        goto resume;
    }
    for (x = 1; x <= N; x++)
    {
        if (isOK(matrix, r, c, x))
        {
            matrix[r * N + c] = x;
            // push all the variables to the stack
            // except n and matrix, they are not changing
            ss[sp] = x;
            sp++;
            ss[sp] = r;
            sp++;
            ss[sp] = c;
            sp++;
            // jump to start
            goto start;
        resume:
            matrix[r * N + c] = 0;
        }
    }
    if (sp != 0)
    {
        // popup variables in reverse order
        sp--;
        c = ss[sp];
        sp--;
        r = ss[sp];
        sp--;
        x = ss[sp];

        // return to the code
        goto resume;
    }
}

__global__ void bootKernel(short *answer, short *pSamples, int *nSize, int *ss)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < (*nSize))
        createLatinSquare(&pSamples[tid * N * N], &ss[tid * STACK_FRAME_SIZE], answer);
}
