#include "framework.cuh"

void init()
{
    short *answer_dev = NULL;
    short *pSamples_dev = NULL;
    int *nSize_dev = NULL;
    int *ss = NULL;
    short *answer = NULL;
    int nSize = 0;
    short *pSamples = NULL;
    size_t ss_size;
    size_t sample_size;
    /*size_t prevLimitStack = 0;
    size_t prevLimitHeap = 0;
    size_t newLimitHeap = 0;*/
    hipError_t err;
    int i, j;

    printf("Generating samples...");
    pSamples = samples_init(&nSize);
    printf("%d samples created.\n", nSize);

    /*
    puts("Configuring device limits...");
    hipDeviceGetLimit(&prevLimitStack, hipLimitStackSize);
    printf("    Current stack limit is %lu\n", prevLimitStack);
    hipDeviceGetLimit(&prevLimitHeap, hipLimitMallocHeapSize);
    printf("    Previous heap limit was %lu\n", prevLimitHeap);
    if (prevLimitHeap >= DEVICE_HEAP_LIMIT)
    {
        puts("Heap limit is big enough, did not change anything.");
    }
    else
    {
        printf("Setting device limit to %lu\n", DEVICE_HEAP_LIMIT);
        err = hipDeviceSetLimit(hipLimitMallocHeapSize, DEVICE_HEAP_LIMIT);
        CHECK_CUDA(err);
        hipDeviceGetLimit(&newLimitHeap, hipLimitMallocHeapSize);
        printf("    Current heap limit is %lu\n", newLimitHeap);
    }
*/
    puts("Allocating Device Memory...");
    ss_size = nSize * STACK_FRAME_SIZE * sizeof(int);
    sample_size = N * N * nSize * sizeof(short);
    printf("Total memory required: %lu B\n", sizeof(int) + ss_size + sample_size + N * N * sizeof(short));
    hipMalloc((void **)&ss, ss_size);
    hipMalloc((void **)&nSize_dev, sizeof(int));
    hipMalloc((void **)&pSamples_dev, sample_size);
    hipMalloc((void **)&answer_dev, N * N * sizeof(short));
    hipMemcpy(pSamples_dev, pSamples, sample_size, hipMemcpyHostToDevice);
    hipMemcpy(nSize_dev, &nSize, sizeof(int), hipMemcpyHostToDevice);
    free(pSamples);
    err = hipGetLastError();
    CHECK_CUDA(err);

    puts("Kernel booting...");
    /*hipStream_t streams[16];
    const int streamLen = 16;
    for (int i = 0; i < streamLen; i++)
    {
        hipStreamCreate(&streams[i]);
        bootKernel<<<1, 1, 0, streams[i]>>>(answer_dev, &pSamples_dev[i * N * N], nSize_dev, &ss[i * STACK_FRAME_SIZE]);
    }*/
    bootKernel<<<512, 256>>>(answer_dev, pSamples_dev, nSize_dev, ss);
    err = hipDeviceSynchronize();
    CHECK_CUDA(err);

    puts("Finalizing...");
    answer = (short *)calloc(N * N, sizeof(short));
    CHECK_NULL(answer);
    hipMemcpy(answer, answer_dev, sizeof(short) * N * N, hipMemcpyDeviceToHost);
    hipFree(answer_dev);
    hipFree(pSamples_dev);
    hipFree(nSize_dev);
    hipFree(ss);
    hipDeviceReset();
    err = hipGetLastError();
    CHECK_CUDA(err);

    puts("Computation done.\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            printf("%d ", answer[i * N + j]);
        }
        putchar('\n');
    }
    free(answer);
}
