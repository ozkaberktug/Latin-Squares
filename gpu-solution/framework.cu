#include "hip/hip_runtime.h"
#include "framework.cuh"

#define INDEXOF(___block___, ___row___, ___col___) (___block___ * n * n + ___row___ * n + ___col___)
#define TOTAL_HEAP_SIZE (STACK_SIZE * blocks + n * n * blocks * sizeof(int) + 1024)

__device__ int getNextEmptyLocation(int *matrix, int n, int *loc_row, int *loc_col)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            if (matrix[i * n + j] == 0)
            {
                *loc_row = i;
                *loc_col = j;
                return 0;
            }
        }
    }
    return 1;
}

__device__ int isOK(int *matrix, int n, int r, int c, int val)
{
    int i;
    for (i = 0; i < n; i++)
    {
        if (matrix[r * n + i] == val)
            return 0;
        if (matrix[i * n + c] == val)
            return 0;
    }
    return 1;
}

__device__ void createLatinSquare(int *matrix, int n, int *ss, int *answer)
{
    // create a stack
    int sp = 0;

    int x, r, c, retVal;

// entry point
start:
    retVal = getNextEmptyLocation(matrix, n, &r, &c);
    if (retVal == 1)
    {
        // call test_function
        /*if (test_function(matrix, n, answer))
            return;*/

        // popup variables in reverse order
        c = ss[--sp];
        r = ss[--sp];
        x = ss[--sp];

        // return to the code
        goto resume;
    }
    for (x = 1; x <= n; x++)
    {
        if (isOK(matrix, n, r, c, x))
        {
            matrix[r * n + c] = x;
            // push all the variables to the stack
            // except n and matrix, they are not changing
            ss[sp++] = x;
            ss[sp++] = r;
            ss[sp++] = c;
            // jump to start
            goto start;
        resume:
            matrix[r * n + c] = 0;
        }
    }
    if (sp != 0)
    {
        // popup variables in reverse order
        c = ss[--sp];
        r = ss[--sp];
        x = ss[--sp];

        // return to the code
        goto resume;
    }

    return;
}

__global__ void runKernel(int *param, int *n, int *blocks, int *ss, int *answer_dev)
{
    if (blockIdx.x < *blocks)
        createLatinSquare(&param[blockIdx.x * (*n) * (*n)], *n, &ss[blockIdx.x * STACK_SIZE], answer_dev);
}

void init(int n)
{
    int i, j;
    hipError_t err;

    puts("Allocating Host Memory...");

    int *answer = (int *)malloc(sizeof(int) * n * n);
    if (answer == NULL)
    {
        fprintf(stderr, "Could not allocate the memory.");
        exit(EXIT_FAILURE);
    }

    const int blocks = fact(n);
    int *param = (int *)malloc(n * n * blocks * sizeof(int));
    if (param == NULL)
    {
        fprintf(stderr, "Could not allocate the memory.");
        exit(EXIT_FAILURE);
    }
    int *array = (int *)malloc(n * sizeof(int));
    if (array == NULL)
    {
        fprintf(stderr, "Could not allocate the memory.");
        exit(EXIT_FAILURE);
    }
    for (i = 0; i < n; i++)
    {
        array[i] = i + 1;
    }
    for (i = 0; i < blocks; i++)
    {
        for (j = 0; j < n; j++)
        {
            int index = INDEXOF(i, 0, j);
            param[index] = array[j];
        }
        nextpermutation(array, n);
    }
    free(array);

/*
    puts("Configuring device limits...");

    size_t limitStack = 0;
    size_t limitHeap = 0;
    hipDeviceGetLimit(&limitStack, hipLimitStackSize);
    printf("    Current stack limit is %lu\n", limitStack);
    hipDeviceGetLimit(&limitHeap, hipLimitMallocHeapSize);
    printf("    Previous heap limit was %lu\n", limitHeap);

    err = hipDeviceSetLimit(hipLimitMallocHeapSize, TOTAL_HEAP_SIZE);
    if (err != hipSuccess)
    {
        fprintf(stderr, "%d:%s\n", err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    size_t newLimitHeap = 0;
    hipDeviceGetLimit(&newLimitHeap, hipLimitMallocHeapSize);
    printf("    Current heap limit is %lu\n", newLimitHeap);
*/
    puts("Allocating Device Memory...");

    int *answer_dev = NULL;
    int *param_dev = NULL;
    int *n_dev = NULL;
    int *blocks_dev = NULL;
    int *ss = NULL;
    hipMalloc((void **)&ss, STACK_SIZE * blocks);
    hipMalloc((void **)&n_dev, sizeof(int));
    hipMalloc((void **)&blocks_dev, sizeof(int));
    hipMalloc((void **)&param_dev, n * n * blocks * sizeof(int));
    hipMalloc((void **)&answer_dev, n * n * sizeof(int));
    hipMemcpy(param_dev, param, n * n * blocks * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(n_dev, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(blocks_dev, &blocks, sizeof(int), hipMemcpyHostToDevice);

    // check for errors:
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "%d:%s\n", err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    puts("Running Kernel...");

    runKernel<<<1, 1>>>(param_dev, n_dev, blocks_dev, ss, answer_dev);

    free(param);
    hipDeviceSynchronize();

    puts("Finalizing...");

    hipMemcpy(answer, answer_dev, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    hipFree(answer_dev);
    hipFree(param_dev);
    hipFree(n_dev);
    hipFree(blocks_dev);
    hipFree(ss);

    // check for errors:
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "%d:%s\n", err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // print results
    puts("Computation done.\n");
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%d ", answer[i * n + j]);
        }
        putchar('\n');
    }
    free(answer);
}
