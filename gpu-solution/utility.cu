
#include <hip/hip_runtime.h>
void swap(int *a, int *b)
{
    if (*a == *b)
        return;
    *a ^= *b;
    *b ^= *a;
    *a ^= *b;
}

void rev(int *s, int l, int r)
{
    while (l < r)
        swap(&s[l++], &s[r--]);
}

int binary_search(int *s, int l, int r, int key)
{
    int index = -1;
    while (l <= r)
    {
        int mid = l + (r - l) / 2;
        if (s[mid] <= key)
            r = mid - 1;
        else
        {
            l = mid + 1;
            if (index == -1 || s[index] >= s[mid])
                index = mid;
        }
    }
    return index;
}

int nextpermutation(int *s, int n)
{
    int len = n, i = len - 2;
    while (i >= 0 && s[i] >= s[i + 1])
        --i;
    if (i < 0)
        return 0;
    else
    {
        int index = binary_search(s, i + 1, len - 1, s[i]);
        swap(&s[i], &s[index]);
        rev(s, i + 1, len - 1);
        return 1;
    }
}

int fact(int n) { return (n <= 1) ? (1) : (n * fact(n - 1)); }
